#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "smart_array.cuh"
#include "peek.cuh"

#define N 32   // test input size

int main() {
    keytype* h_keys = (keytype*)malloc(N * sizeof(keytype));
    valuetype* h_values = (valuetype*)malloc(N * sizeof(valuetype));

    srand(0);
    // Initialize the input and sorted arrays
    for (int i = 0; i < N; i++) {
        h_keys[i] = rand() % (N / 2); // limiting the values to ensure key repetition
    }
    keytype* d_keys;
    valuetype* d_values;

    // --------------------------- allocating GPU memory ---------------------------
    hipMalloc(&d_keys, N * sizeof(keytype));
    hipMalloc(&d_values, N * sizeof(valuetype));

    // --------------------------- Moving data from RAM to GPU memory ---------------------------
    hipMemcpy(d_keys, h_keys, N * sizeof(keytype), hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, N * sizeof(valuetype), hipMemcpyHostToDevice);

    MagicArray arr(10000);
    arr.insert(d_keys, d_values, N);
    // arr.printTable();

    free(h_keys);
    free(h_values);
    hipFree(d_keys);
    hipFree(d_values);

    return 0;
}
