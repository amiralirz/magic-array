#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/reduce.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#define N 100000
#define MAX_OCCURRENCES 1000  // Max occurrences of each key

struct KeyValuePair {
    int key;
    int value_index;  // Store index of value instead of value itself
};

typedef struct {
    int key;
    int indices[MAX_OCCURRENCES];  // Indices of values
} AggregatedElement;

__global__ void aggregateIndices(int *keys, int *indices, AggregatedElement *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int key = keys[idx];
        int index = indices[idx];
        for (int i = 0; i < size; i++) {
            if (atomicCAS(&output[i].key, -1, key) == -1 || output[i].key == key) {
                for (int j = 0; j < MAX_OCCURRENCES; j++) {
                    if (atomicCAS(&output[i].indices[j], -1, index) == -1) {
                        return;
                    }
                }
            }
        }
    }
}

int main() {
    // Seed random number generator
    std::srand(std::time(0));

    // Create large array of keys with repeated keys
    int h_keys[N];
    int h_indices[N];
    for (int i = 0; i < N; i++) {
        h_keys[i] = std::rand() % (N / 100 + 1); // Repeat keys
        h_indices[i] = i;
    }

    // Transfer to device
    thrust::device_vector<int> d_keys(h_keys, h_keys + N);
    thrust::device_vector<int> d_indices(h_indices, h_indices + N);

    // Sort by keys
    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

    // Allocate output vector
    thrust::device_vector<AggregatedElement> d_output(N);
    AggregatedElement empty_element;
    empty_element.key = -1;
    for (int i = 0; i < MAX_OCCURRENCES; i++) empty_element.indices[i] = -1;
    thrust::fill(d_output.begin(), d_output.end(), empty_element);

    // Launch kernel to aggregate indices
    aggregateIndices<<<(N + 255) / 256, 256>>>(thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_indices.data()), thrust::raw_pointer_cast(d_output.data()), N);
    hipDeviceSynchronize(); // Ensure kernel has completed

    // Sort the output by keys on the device
    // thrust::sort(thrust::device, d_output.begin(), d_output.end(), [] __device__ (const AggregatedElement &a, const AggregatedElement &b) {
    //     return a.key < b.key;
    // });

    printf("here\n");

    // Copy back to host to print results
    std::vector<AggregatedElement> h_output(N);
    thrust::copy(d_output.begin(), d_output.end(), h_output.begin());

    printf("there\n");

    // Print results (limited to first 10 for brevity)
    for (const auto &element : h_output) {
        if (element.key == -1) break;
        std::cout << "Key: " << element.key << " -> Indices: ";
        for (int i = 0; i < MAX_OCCURRENCES; i++) {
            if (element.indices[i] != -1) std::cout << element.indices[i] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
